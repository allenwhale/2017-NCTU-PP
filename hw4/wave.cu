/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", 
                    MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__device__ inline unsigned getIdx(){
    return blockIdx.x * blockDim.x + threadIdx.x;
}
__global__ void update(float *values, int nsteps, int tpoints)
{
    float local_value, local_old_value, local_new_value;
    int idx = getIdx();
    local_old_value = local_value = sin((2.0 * PI) * ((float)idx / (float)(tpoints - 1)));
    for (int i = 0 ; i < nsteps; i++) {
        local_new_value = (1.82 * local_value) - local_old_value;
        local_old_value = local_value;
        local_value = local_new_value;
    }
    if(0 < idx && idx < tpoints - 1){
        values[idx] = local_value;
    }else if(idx == 0 || idx == tpoints - 1){
        values[idx] = 0;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 0; i < tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i % 10 == 9)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    printf("Initializing points on the line...\n");
    printf("Updating all points for all time steps...\n");
    float *d_values;
    hipMalloc(&d_values, sizeof(values));
    update<<<((tpoints + 1023) >> 10), 1024>>>(d_values, nsteps, tpoints);
    hipMemcpy(values, d_values, sizeof(values), hipMemcpyDeviceToHost);
    printf("Printing final results...\n");
    printfinal();
    hipFree(d_values);
    printf("\nDone.\n\n");

    return 0;
}
